#include "include/cuda_lib.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C" void cuda_init_runtime_on_primary() { (void)hipFree(0); }

extern "C" int bind_primary_ctx(int dev_ord) {
  hipDevice_t d;
  hipCtx_t ctx;
  if (hipInit(0))
    return 1;
  if (hipDeviceGet(&d, dev_ord))
    return 2;
  if (hipDevicePrimaryCtxRetain(&ctx, d))
    return 3;
  if (hipCtxSetCurrent(ctx))
    return 4;
  (void)hipFree(0); // attach runtime to this context on this thread
  return 0;
}
